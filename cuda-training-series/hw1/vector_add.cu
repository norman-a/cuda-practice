
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
    
const int N = 4096;
const int thread_size = 256;  // CUDA maximum is 1024
__global__ void add_vector(int* a, int* b, int* c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}


void random_int(int* vector, int size){
    for (int i = 0; i < size; i++){
        vector[i] = rand() % 10;
    }
}
int main(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    srand(time(NULL));
    a = new int[N];
    b = new int[N];
    c = new int[N];

    random_int(a, N);
    random_int(b, N);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy failure");


    add_vector<<<(N + thread_size - 1) / thread_size, thread_size>>>(d_a, d_b, d_c);
    cudaCheckErrors("kernel launch failure");
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("A,   B,   C\n");
    for (int i = 0; i < N; i++){
        printf("%d    %d    %d\n", a[i], b[i], c[i]);
    }
    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);
    free(c);
    hipFree(d_c);
    return 0;
}